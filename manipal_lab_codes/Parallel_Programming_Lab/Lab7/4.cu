
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
// #include <math.h>

__global__ void func(double *angle, double *sine){
	int idx = threadIdx.x;
	sine[idx] = sin(angle[idx]);
}

int main()
{
	double *in, *out;
	int n;
	printf("Enter the number of elements: ");
	scanf("%d",&n);

	in = (double *)malloc(n*sizeof(double));
	out = (double *)malloc(n*sizeof(double));
	
	printf("Enter the elements of the angle vector: ");
	for(int i=0; i<n; i++){
		scanf("%lf", &in[i]);
	}

	double *dIn, *dOut;
	
	int size = n*sizeof(double);

	hipMalloc((void **)&dIn,size);
	hipMalloc((void **)&dOut,size);

	hipMemcpy(dIn, in, size, hipMemcpyHostToDevice);
	hipMemcpy(dOut, out, size, hipMemcpyHostToDevice);

	func<<<1,n>>>(dIn, dOut);

	hipMemcpy(out,dOut,size,hipMemcpyDeviceToHost);

	printf("Angles and their sines are:\n");
	for(int i=0; i<n; i++){
		printf("%0.4lf\t%0.4lf\n", in[i], out[i]);
	}
	printf("\n");

	hipFree(dIn);
	hipFree(dOut);

	return 0;
}
