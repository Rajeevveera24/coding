
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void func(int *y, int *x, int *a)
{
	int idx = threadIdx.x;
	y[idx] = *a * x[idx] + y[idx];
}

int main()
{
	int *x, *y, a;
	int n;
	printf("Enter the number of elements: ");
	scanf("%d",&n);

	x = (int *)malloc(n*sizeof(int));
	y = (int *)malloc(n*sizeof(int));
	// C = (int *)malloc(N*sizeof(int));
	// D = (int *)malloc(N*sizeof(int));
	// E = (int *)malloc(N*sizeof(int));
	
	printf("Enter the elements of the vector X: ");
	for(int i=0; i<n; i++){
		scanf("%d", &x[i]);
	}
	
	printf("Enter the elements of the vector Y: ");
	for(int i=0; i<n; i++){
		scanf("%d", &y[i]);
	}

	printf("Enter the value of a: ");
	scanf("%d", &a);

	int *dX, *dY, *dA;
	
	int size = n*sizeof(int);

	hipMalloc((void **)&dX,size);
	hipMalloc((void **)&dY,size);
	hipMalloc((void **)&dA,1);

	hipMemcpy(dY, y, size, hipMemcpyHostToDevice);
	hipMemcpy(dX, x, size, hipMemcpyHostToDevice);
	hipMemcpy(dA, &a, 1, hipMemcpyHostToDevice);

	func<<<1,n>>>(dY, dX, dA);

	hipMemcpy(y,dY,size,hipMemcpyDeviceToHost);

	printf("Vector Y has the values: ");
	for(int i=0; i<n; i++){
		printf("%d ",y[i]);
	}
	printf("\n");

	hipFree(dX);
	hipFree(dY);
	hipFree(dA);

	return 0;
}
