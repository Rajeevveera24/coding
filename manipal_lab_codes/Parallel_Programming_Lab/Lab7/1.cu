
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void add_vector_block_N(int *A, int *B, int *C)
{
	int idx = blockIdx.x;
	C[idx] = A[idx] + B[idx];
}

__global__ void add_vector_thread_N(int *A, int *B, int *C)
{
	int idx = threadIdx.x;
	C[idx] = A[idx] + B[idx];
}

__global__ void add_vector_variable_N(int *A, int *B, int *C, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx<N){
		C[idx] = A[idx] + B[idx];
	}
}

int main()
{
	int *A,*B,*C,*D,*E;
	int N;
	printf("Enter the number of elements: ");
	scanf("%d",&N);

	A = (int *)malloc(N*sizeof(int));
	B = (int *)malloc(N*sizeof(int));
	C = (int *)malloc(N*sizeof(int));
	D = (int *)malloc(N*sizeof(int));
	E = (int *)malloc(N*sizeof(int));
	
	printf("Enter the elements of the first vector: ");
	for(int i=0;i<N;i++){
		scanf("%d", &A[i]);
	}
	printf("Enter the elements of the second vector: ");
	for(int i=0;i<N;i++){
		scanf("%d", &B[i]);
	}
	int *d_a,*d_b,*d_c,*d_d,*d_e;
	
	int size = N*sizeof(int);

	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_c,size);
	hipMalloc((void **)&d_d,size);
	hipMalloc((void **)&d_e,size);

	hipMemcpy(d_a, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, B, size, hipMemcpyHostToDevice);

	add_vector_block_N<<<N,1>>>(d_a,d_b,d_c);

	hipMemcpy(C,d_c,size,hipMemcpyDeviceToHost);

	printf("Block size of N: ");
	for(int i=0;i<N;i++){
		printf("%d ",C[i]);
	}
	printf("\n");

	add_vector_thread_N<<<1,N>>>(d_a,d_b,d_d);

	hipMemcpy(D,d_d,size,hipMemcpyDeviceToHost);

	printf("N threads: ");
	for(int i=0;i<N;i++){
		printf("%d ",C[i]);
	}
	printf("\n");

	add_vector_variable_N<<<ceil(N/256.0),256>>>(d_a, d_b, d_c, N);

	hipMemcpy(E, d_e, size, hipMemcpyDeviceToHost);

	printf("Variable size blocks: ");
	for(int i=0;i<N;i++){
		printf("%d ",C[i]);
	}
	printf("\n");

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipFree(d_d);
	hipFree(d_e);

	return 0;
}
