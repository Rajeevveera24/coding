
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matMulRowByThread(int *a, int *b, int *c, int Wa, int Wb){
	
	int sum;
	int i = threadIdx.x;

	for(int j=0; j<Wb; j++){
		sum = 0;
		for(int k=0; k<Wa; k++){
			sum += a[i*Wa + k] * b[k*Wb + j];
		}
		c[i*Wb + j] = sum;
	}
}

__global__ void matMulColumnByThread(int *a, int *b, int *c, int Hb, int Ha){
	
	int sum;
	int j = threadIdx.x;

	for(int i=0; i<Ha; j++){
		sum = 0;
		for(int k=0; k<Hb; k++){
			sum += a[i*Hb + k] * b[k*blockDim.x + j];
		}
		c[i*blockDim.x + j] = sum;
	}
}

__global__ void matMulElementByThread(int *a, int *b, int *c, int Wa){
	
	int sum;
	int j = threadIdx.x;
	int i = threadIdx.y;
	for(int k=0; k<Wa; k++){
		sum += a[i*Wa + k] * b[k*blockDim.x + j];
	}
	c[i*blockDim.x + j] = sum;
}

int main()
{
	int *A, *B, *C, *D, *E;
	int ha, wa, hb, wb;
	
	printf("Enter the row and column sizes of first Matrix: ");
	scanf("%d %d", &ha, &wa);
	printf("Enter the elements of the matrix\n");
	A = (int *)malloc(ha*wa*sizeof(int));
	for(int i=0; i<ha*wa; i++){
		scanf("%d", &A[i]);
	}

	printf("Enter the row and column sizes of second Matrix: ");
	scanf("%d %d", &hb, &wb);
	printf("Enter the elements of the matrix\n");
	B = (int *)malloc(hb*wb*sizeof(int));
	for(int i=0; i<hb*wb; i++){
		scanf("%d", &B[i]);
	}

	int sizeA = ha * wa * sizeof(int);
	int sizeB = hb * wb * sizeof(int);
	int sizeC = ha * wb * sizeof(int);
	
	dim3 gridDim(1,1,1);
	dim3 blockDim(ha, 1, 1);
	
	C = (int *)malloc(sizeC);
	D = (int *)malloc(sizeC);
	E = (int *)malloc(sizeC);
	
	int *d_a,*d_b,*d_c,*d_d,*d_e;

	hipMalloc((void **)&d_a ,sizeA);
	hipMalloc((void **)&d_b, sizeB);
	hipMalloc((void **)&d_c, sizeC);
	hipMalloc((void **)&d_d, sizeC);
	hipMalloc((void **)&d_e, sizeC);

	hipMemcpy(d_a, A, sizeA, hipMemcpyHostToDevice);
	hipMemcpy(d_b, B, sizeB, hipMemcpyHostToDevice);

	matMulRowByThread <<< gridDim, blockDim >>> (d_a, d_b, d_c, wa, wb);

	hipMemcpy(C,d_c, sizeC, hipMemcpyDeviceToHost);

	printf("Resultant Matrix when each row is computed by one thread:\n");
	for(int i=0; i<ha; i++){
		for(int j=0; j<wb; j++){
			printf("%d ", C[i*ha + j]);
		}
		printf("\n");
	}
	
	dim3 gridDim1(1,1,1);
	dim3 blockDim1(wb, 1, 1);

	hipMemcpy(d_a, A, sizeA, hipMemcpyHostToDevice);
	hipMemcpy(d_b, B, sizeB, hipMemcpyHostToDevice);

	matMulColumnByThread <<< gridDim1, blockDim1 >>> (d_a, d_b, d_d, hb, ha);

	hipMemcpy(D,d_d, sizeC, hipMemcpyDeviceToHost);

	printf("Resultant Matrix when each column is computed by one thread:\n");
	for(int i=0; i<ha; i++){
		for(int j=0; j<wb; j++){
			printf("%d ", D[i*ha + j]);
		}
		printf("\n");
	}

	dim3 gridDim2(1,1,1);
	dim3 blockDim2(wb, ha, 1);

	hipMemcpy(d_a, A, sizeA, hipMemcpyHostToDevice);
	hipMemcpy(d_b, B, sizeB, hipMemcpyHostToDevice);

	matMulElementByThread <<< gridDim2, blockDim2 >>> (d_a, d_b, d_e, wa);

	hipMemcpy(E, d_e, sizeC, hipMemcpyDeviceToHost);

	printf("Resultant Matrix when each element is computed by one thread:\n");
	for(int i=0; i<ha; i++){
		for(int j=0; j<wb; j++){
			printf("%d ", E[i*ha + j]);
		}
		printf("\n");
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipFree(d_d);
	hipFree(d_e);

	free(A);
	free(B);
	free(C);
	free(D);
	free(E);

	return 0;
}

