
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void row_thread(int *A, int *B, int*C,int M, int N){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < M){
		for(int i=0;i<N;i++){
			C[id*N + i] = A[id*N + i] + B[id*N + i];
		}
	}
}

__global__ void col_thread(int *A, int *B, int*C,int M, int N){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < N){
		for(int i=0; i<M; i++){
			C[i*N + id] = A[i*N + id] + B[i*N + id];
		}
	}
}

__global__ void one_each_thread(int *A, int *B, int*C,int M, int N)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < M*N){
		C[id] = A[id] + B[id];
	}
}

int main()
{
	int *A, *B, *C, *D, *E, *d_a, *d_b, *d_c, *d_d, *d_e;
	int M, N, i, j, size;

	printf("Enter the dimensions of matrix : ");
	scanf("%d %d", &M, &N);
	
	size = M * N * sizeof(int);

	A = (int *)malloc(size);
	B = (int *)malloc(size);
	C = (int *)malloc(size);
	D = (int *)malloc(size);
	E = (int *)malloc(size);
	
	printf("Enter matrix A :\n");
	for(i=0; i<M; i++){
		for(j=0; j<N; j++){
			scanf("%d", &A[i*N + j]);
		}
	}
	
	printf("Enter matrix B :\n");
	for(i=0; i<M; i++){
		for(j=0; j<N; j++){
			scanf("%d", &B[i*N + j]);
		}
	}

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	hipMalloc((void **)&d_d, size);
	hipMalloc((void **)&d_e, size);

	hipMemcpy(d_a, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, B, size, hipMemcpyHostToDevice);

	row_thread <<< 1, M >>> (d_a, d_b, d_c, M, N);
	col_thread <<< 1, N >>> (d_a, d_b, d_d, M, N);
	one_each_thread <<< 1, M*N>>> (d_a, d_b, d_e, M, N);

	hipMemcpy(C, d_c, size, hipMemcpyDeviceToHost);
	hipMemcpy(D, d_d, size, hipMemcpyDeviceToHost);
	hipMemcpy(E, d_e, size, hipMemcpyDeviceToHost);

	printf("Result for one thread each row :\n");
	for(i=0;i<M;i++){
		for(j=0;j<N;j++){
			printf("%d ",C[i*N + j]);
		}
		printf("\n");
	}

	printf("Result for one thread each col :\n");
	for(i=0;i<M;i++){
		for(j=0;j<N;j++){
			printf("%d ",D[i*N + j]);
		}
		printf("\n");
	}

	printf("Result for one thread each element :\n");
	for(i=0;i<M;i++){
		for(j=0;j<N;j++){
			printf("%d ",E[i*N + j]);
		}
		printf("\n");
	}
}
