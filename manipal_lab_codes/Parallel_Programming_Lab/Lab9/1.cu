
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void evaluate_power(int *A, int M, int N){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	for(int i=0;i<N;i++){
		A[id * N + i] = powf(A[id * N + i], id+1);
	}
}

int main()
{
	int *A, *d_a;
	int M, N, size;

	printf("Enter the dimensions of matrix : ");
	
	scanf("%d %d", &M, &N);
	
	A = (int *)malloc(sizeof(int) * M * N);
	
	printf("Enter matrix elements :\n");
	for(int i=0; i<M*N; i++){
		scanf("%d",&A[i]);
	}

	size = sizeof(int) * M * N;

	hipMalloc((void **)&d_a, size);

	hipMemcpy(d_a, A, size, hipMemcpyHostToDevice);

	evaluate_power<<<ceil(M/256.0), M>>>(d_a, M, N);

	hipMemcpy(A, d_a, size, hipMemcpyDeviceToHost);

	printf("Modified Matrix:\n");
	for(int i=0;i<M;i++){
		for(int j=0;j<N;j++){
			printf("%d ",A[i*N + j]);
		}
		printf("\n");
	}

	hipFree(d_a);

	free(A);

	return 0;
}
